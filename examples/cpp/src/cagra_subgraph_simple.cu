/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdint>
#include <raft/core/handle.hpp>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/random/make_blobs.cuh>

#include <cuvs/neighbors/cagra.hpp>
#include <cuvs/neighbors/brute_force.hpp>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include "common.cuh"

#define CATS 5

struct cagra_filter {
  raft::device_vector_view<int32_t> data_cats;
  raft::device_vector_view<int32_t> query_cats;

  inline _RAFT_HOST_DEVICE bool operator()(
    // query index
    const uint32_t query_ix,
    const uint32_t sample_ix) const
  {
    return data_cats(sample_ix) == query_cats(query_ix);
  }
};


// Fill dataset and queries with synthetic data.
void generate_dataset_test(raft::device_resources const &dev_resources,
                      raft::device_matrix_view<float, int64_t> dataset,
                      raft::device_matrix_view<float, int64_t> queries,
                      int n_clusters,
                      int64_t seed) {
  auto labels = raft::make_device_vector<int64_t, int64_t>(dev_resources,
                                                           dataset.extent(0));
  hipStream_t stream = raft::resource::get_cuda_stream(dev_resources);
  raft::random::make_blobs<float,int64_t>(dataset.data_handle(),
                           labels.data_handle(),
                           dataset.extent(0),
                           dataset.extent(1),
                           n_clusters,
                           stream, 
                           true,
                           nullptr,
                           nullptr,
                           float(1.0),
                           false,
                           (float)-10.0f,
                           (float)10.0f,
                           (uint64_t)seed);

  raft::random::RngState r(seed);
  raft::random::uniform(
      dev_resources, r,
      raft::make_device_vector_view(queries.data_handle(), queries.size()),
      -10.0f, 10.0f);
}

// Generate random categories for each data vector and query
void generate_categories(raft::device_resources const& dev_resources,
                      raft::device_vector_view<int32_t> dataset,
                      raft::device_vector_view<int32_t> queries,
                      int num_categories)
{
  raft::random::RngState r(1234ULL);
  raft::random::uniformInt(dev_resources, r, raft::make_device_vector_view(dataset.data_handle(), dataset.size()), 0,num_categories);
  raft::random::uniformInt(dev_resources, r, raft::make_device_vector_view(queries.data_handle(), queries.size()), 0,num_categories);

}

void compute_recall_unfiltered(raft::device_resources const& dev_resources, 
                    raft::device_matrix_view<const float, int64_t> dataset, 
                    raft::device_matrix_view<const float, int64_t> queries,
                    raft::device_matrix_view<uint32_t> neighbors,
                    int topk) {

  int n_queries = queries.extent(0);
  hipStream_t stream = raft::resource::get_cuda_stream(dev_resources);

  auto bf_index = cuvs::neighbors::brute_force::build(dev_resources, dataset);
  
  auto bf_neighbors = raft::make_device_matrix<int64_t,int64_t>(dev_resources, n_queries, topk);
  auto bf_distances = raft::make_device_matrix<float,int64_t>(dev_resources, n_queries, topk);

  cuvs::neighbors::brute_force::search(dev_resources, bf_index, queries, bf_neighbors.view(), bf_distances.view(), std::nullopt);

  auto h_bf_neighbors = raft::make_host_matrix<int64_t,int64_t>(n_queries, topk);
  raft::copy(h_bf_neighbors.data_handle(), bf_neighbors.data_handle(), bf_neighbors.size(), stream);

  auto h_neighbors = raft::make_host_matrix<uint32_t>(n_queries, topk);
  raft::copy(h_neighbors.data_handle(), neighbors.data_handle(), neighbors.size(), stream);

//printf("computing unfiltered recall...\n");
  int correct=0;
  for(int i=0; i<neighbors.extent(0); i++) {
    for(int j=0; j<topk; j++) {
      for(int k=0; k<topk; k++) {
        if(h_neighbors(i,j) == h_bf_neighbors(i,k)) {
          correct++;
          break;
        }
      }
    }
  }

  printf("Recall - queries:%d, topk:%d, correct:%d, recall:%f\n", n_queries, topk, correct, (float)correct / (float)(n_queries*topk));

   
}

void cagra_build_search_subgraphs(raft::device_resources const& dev_resources,
                               std::vector<raft::device_matrix<float, int64_t>> datasets,
                               std::vector<raft::device_matrix<float, int64_t>> queries,
                               int n_categories,
                               int topk)
{
  printf("Starting fcn...\n");
  using namespace cuvs::neighbors;

  hipStream_t stream = raft::resource::get_cuda_stream(dev_resources);

  // Create neighbors and dist output lists for each cat
  std::vector<raft::device_matrix<uint32_t, int64_t>> neighbors;
  neighbors.reserve(n_categories);
  std::vector<raft::device_matrix<float,int64_t>> distances;
  distances.reserve(n_categories);

  // use default index parameters
  cagra::index_params index_params;

  std::vector<cuvs::neighbors::cagra::index<float,uint32_t>> indexes;
  indexes.reserve(n_categories);

  for(int i=0; i<n_categories; i++) {
    neighbors.push_back(raft::make_device_matrix<uint32_t,int64_t>(dev_resources, queries[i].extent(0), topk));
    distances.push_back(raft::make_device_matrix<float,int64_t>(dev_resources, queries[i].extent(0), topk));

    printf("calling build for dataset %d...\n", i);

    indexes.push_back(cagra::build(dev_resources, index_params, raft::make_const_mdspan(datasets[i].view())));
  }

  raft::resource::sync_stream(dev_resources);

  printf("Indexes all created!\n");

  // use default search parameters
  cagra::search_params search_params;
  // search K nearest neighbors
  for(int i=0; i<n_categories; i++) {
    cagra::search(dev_resources, search_params, indexes[i], raft::make_const_mdspan(queries[i].view()), neighbors[i].view(), distances[i].view());

//    print_results(dev_resources, neighbors[i].view(), distances[i].view());
    printf("Label: %d - ", i);
    compute_recall_unfiltered(dev_resources, datasets[i].view(), queries[i].view(), neighbors[i].view(), topk);
  }

}

void print_vecs(raft::device_resources const &dev_resources, 
                  raft::device_matrix_view<float,int64_t> dataset,
                  raft::device_matrix_view<float,int64_t> queries) {
  auto h_data = raft::make_host_matrix<float,int64_t>(dataset.extent(0), dataset.extent(1));
  auto h_queries = raft::make_host_matrix<float,int64_t>(queries.extent(0), queries.extent(1));

  hipStream_t stream = raft::resource::get_cuda_stream(dev_resources);
  raft::copy(h_data.data_handle(), dataset.data_handle(), dataset.size(), stream);
  raft::copy(h_queries.data_handle(), queries.data_handle(), queries.size(), stream);

  printf("data:\n");
  for(int i=0; i<10; i++) {
    printf("i:%d - ", i);
    for(int j=0; j<h_data.extent(1); j++) {
      printf("%f, ", h_data(i,j));
    }
    printf("\n");
  }
  printf("queries:\n");
  for(int i=0; i<10; i++) {
    printf("i:%d - ", i);
    for(int j=0; j<h_queries.extent(1); j++) {
      printf("%f, ", h_queries(i,j));
    }
    printf("\n");
  }
}

int main()
{
  raft::device_resources dev_resources;

  // Set pool memory resource with 1 GiB initial pool size. All allocations use the same pool.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr(
    rmm::mr::get_current_device_resource(), 10*1024 * 1024 * 1024ull);
  rmm::mr::set_current_device_resource(&pool_mr);

  // Create input arrays.
  int64_t n_samples = 10000;
  int64_t n_dim     = 5;
  int64_t n_queries = 100;
  int64_t topk      = 10;
  int n_categories = CATS;
  int n_data_clusters = 10;
  std::vector<raft::device_matrix<float, int64_t>> datasets;
  datasets.reserve(n_categories);
  std::vector<raft::device_matrix<float, int64_t>> queries;
  queries.reserve(n_categories);

  for(int i=0; i<n_categories; i++) {
    datasets.push_back(raft::make_device_matrix<float, int64_t>(dev_resources, n_samples, n_dim));
    queries.push_back(raft::make_device_matrix<float, int64_t>(dev_resources, n_queries, n_dim));
    generate_dataset_test(dev_resources, datasets[i].view(), queries[i].view(), n_data_clusters, i);

  }

//  for(int i=0; i<n_categories; i++) {
//    print_vecs(dev_resources, datasets[i].view(), queries[i].view());
//  }

    // Simple build and search example.
  cagra_build_search_subgraphs(dev_resources,
                            datasets, queries, n_categories, topk);

  return 0;
}
